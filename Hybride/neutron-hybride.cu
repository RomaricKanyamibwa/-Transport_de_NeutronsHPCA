/*
 * Université Pierre et Marie Curie
 * Calcul de transport de neutrons
 * Version séquentielle
 */

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <omp.h>

#include "device_atomic_functions.h"

#define OUTPUT_FILE "/tmp/romhar/absorbed.dat"
#define THREAD_PER_BLOCK 256 
#define NB_BLOCKS 256

char info[] = "\
Usage:\n\
    neutron-seq H Nb C_c C_s\n\
\n\
    H  : épaisseur de la plaque\n\
    Nb : nombre d'échantillons\n\
    C_c: composante absorbante\n\
    C_s: componente diffusante\n\
\n\
Exemple d'execution : \n\
    neutron-seq 1.0 500000000 0.5 0.5\n\
";

/*
 * notre gettimeofday()
 */
double my_gettimeofday(){
  struct timeval tmp_time;
  gettimeofday(&tmp_time, NULL);
  return tmp_time.tv_sec + (tmp_time.tv_usec * 1.0e-6L);
}

__global__ void setup_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x*gridDim.x;
    hiprand_init(16453, id, 0, &state[id]);
}

__global__ void neutron_calculus(hiprandState *state, float c, float c_c, float h, float* absorbed, int* result, int n, int* c_abs){
    int id = threadIdx.x + blockIdx.x*blockDim.x;
    int pos_ecrit;
    int pos_Thread = id;
    __shared__ int r[THREAD_PER_BLOCK];
    __shared__ int b[THREAD_PER_BLOCK];
    __shared__ int t[THREAD_PER_BLOCK];
    r[threadIdx.x] = 0;
    t[threadIdx.x] = 0;
    b[threadIdx.x] = 0;
    float L;
    float u;
    float d;
    float x;
    while(pos_Thread < n) {
	      d = 0.0;
              x = 0.0;
              while (1) {
	      u = hiprand_uniform (&state[id]);
	      L = -(1 / c) * log(u);
	      x = x + L * cos(d);
	      if (x < 0) {
		r[threadIdx.x] = r[threadIdx.x]+1;
		break;
	      } else if (x >= h) {
		t[threadIdx.x] = t[threadIdx.x]+1;
		break;
	      } else if ((u = hiprand_uniform (&state[id])) < c_c / c) {
		
		b[threadIdx.x] = b[threadIdx.x]+1;
		pos_ecrit = atomicAdd(c_abs, 1);
		absorbed[pos_ecrit] = x;
		
		break;
	      } else {
		u = hiprand_uniform (&state[id]);
		d = u * M_PI;
	      }
	    }
	pos_Thread = pos_Thread + gridDim.x*blockDim.x;
	}
	__syncthreads();
	int j = blockDim.x/2;
	while(j>0){
		if(threadIdx.x<j){
			r[threadIdx.x] += r[threadIdx.x + j];
			t[threadIdx.x] += t[threadIdx.x + j];
			b[threadIdx.x] += b[threadIdx.x + j];
		}
		j/=2;
		__syncthreads();
	}
	if(threadIdx.x==0){
		atomicAdd(result,r[0]);
		atomicAdd(result+1,t[0]);
		atomicAdd(result+2,b[0]);
	}
}


int main(int argc, char *argv[]) {
  // La distance moyenne entre les interactions neutron/atome est 1/c. 
  // c_c et c_s sont les composantes absorbantes et diffusantes de c. 
  float c, c_c, c_s;
  // épaisseur de la plaque
  float h;
  // nombre d'échantillons
  int n;
  // nombre de neutrons refléchis, absorbés et transmis
  int* result = (int *) calloc(3, sizeof(int)); //r, t, b
  // chronometrage
  double start, finish;
	// variable threads openMP
	int tid, nthreads;

  int j = 0; // compteurs 
  
  //perf files
  FILE *perf = fopen("../perform.txt", "a+");
  FILE *perf_gnuplot = fopen("../perform_gnuplot.txt", "a+");
  char str[512];

  if( argc == 1)
    fprintf( stderr, "%s\n", info);

  // valeurs par defaut
  h = 1.0;
  n = 500000000;
  c_c = 0.5;
  c_s = 0.5;

  // recuperation des parametres
  if (argc > 1)
    h = atof(argv[1]);
  if (argc > 2)
    n = atoi(argv[2]);
  if (argc > 3)
    c_c = atof(argv[3]);
  if (argc > 4)
    c_s = atof(argv[4]);
  c = c_c + c_s;

  // affichage des parametres pour verificatrion
  printf("Épaisseur de la plaque : %4.g\n", h);
  printf("Nombre d'échantillons  : %d\n", n);
  printf("C_c : %g\n", c_c);
  printf("C_s : %g\n", c_s);
  printf("Thread per block : %d\n",THREAD_PER_BLOCK);
  printf("Number of block : %d\n",NB_BLOCKS);


  float *absorbed;
  absorbed = (float *) calloc(n, sizeof(float));
  int nb_thread = 256;
  dim3 threadsParBloc(nb_thread,1,1);
  dim3 nbBlocks(256,1,1);
  float* absorbed_gpu;
  int* result_gpu;
  int* c_abs;
  hiprandState* d_state;
  
	// DEBUT CHRONO
  start = my_gettimeofday();

	// MEMOIRE GPU RESERVATION 
  hipMalloc(&d_state, nb_thread*nbBlocks.x*sizeof(hiprandState));
  hipMalloc(&absorbed_gpu, n*sizeof(float));
  hipMalloc(&result_gpu, 3*sizeof(int));
  hipMalloc(&c_abs, sizeof(int));

	// INITIALISATION VARIABLES LEGERES GPU
  hipMemset(c_abs,0,sizeof(int));
  hipMemset(result_gpu,0,3*sizeof(int));

	// INITIALISATION VARIABLE LOURDE GPU EN PARALLELE
	#pragma omp parallel
	{
		nthreads = omp_get_num_threads();
		tid=omp_get_thread_num();
		hipMemset(absorbed_gpu+tid*n/nthreads,0.0,n*sizeof(float)/nthreads);
        if(tid==0)
            printf("Number of Threads : %d\n",nthreads );
	}

	// CALCUL SUR GPU
  setup_kernel<<<nbBlocks, threadsParBloc >>>(d_state);
  neutron_calculus<<<nbBlocks, threadsParBloc >>>(d_state, c, c_c, h, absorbed_gpu, result_gpu, n, c_abs);

	// RECUPERATION DES VARIABLES LOURDES GPU EN PARALLEL SUR CPU (n*8 octets, par ex 500000000*8 = 4 Go)
	#pragma omp parallel
	{
		nthreads = omp_get_num_threads();
		tid=omp_get_thread_num();
		hipMemcpy(absorbed+tid*n/nthreads, absorbed_gpu+tid*n/nthreads, n*sizeof(float)/nthreads,hipMemcpyDeviceToHost);
	}
	
	//RECUPERATION VARIABLES LEGERES (12 octets)
	hipMemcpy(result, result_gpu, 3*sizeof(int),hipMemcpyDeviceToHost);

	// FIN CHRONO
  finish = my_gettimeofday();

  //printf("\nTemps total de calcul: %.8g sec\n", finish - start);
  sprintf(str,"***************Hybride N:%d ***************\n\
  Nb_thread:%d , Nb_Blocs:%d Omp_num_threads=%d \n\
  #Temps total de calcul : %.8g seconde(s)\n\n"
            ,n,THREAD_PER_BLOCK,NB_BLOCKS,nthreads,finish-start);

	fwrite(str,sizeof(char),strlen(str),perf);
	sprintf(str,"%d %.8g %d %d \n",n,finish-start,THREAD_PER_BLOCK,NB_BLOCKS);
	fwrite(str,sizeof(char),strlen(str),perf_gnuplot);


  int r = result[0];
  int t = result[1];
  int b = result[2];
  printf("\nPourcentage des neutrons refléchis : %4.2g\n", (float) r / (float) n);
  printf("Pourcentage des neutrons absorbés : %4.2g\n", (float) b / (float) n);
  printf("Pourcentage des neutrons transmis : %4.2g\n", (float) t / (float) n);
  printf("Nombre de neutrons traites : %d\n", r+b+t);
  printf("\nTemps total de calcul: %.8g sec\n", finish - start);
  printf("Millions de neutrons /s: %.2g\n", (double) n / ((finish - start)*1e6));

  // ouverture du fichier pour ecrire les positions des neutrons absorbés
  FILE *f_handle = fopen(OUTPUT_FILE, "w");
  if (!f_handle) {
    fprintf(stderr, "Cannot open " OUTPUT_FILE "\n");
    exit(EXIT_FAILURE);
  }

  for (j = 0; j < b; j++)
    fprintf(f_handle, "%f\n", absorbed[j]);

  // fermeture du fichier
  fclose(f_handle);
  printf("Result written in " OUTPUT_FILE "\n"); 

  free(absorbed);

  return EXIT_SUCCESS;
}

