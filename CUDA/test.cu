/*
 * Université Pierre et Marie Curie
 * Calcul de transport de neutrons
 * Version séquentielle
 */

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "device_atomic_functions.h"

#define OUTPUT_FILE "/tmp/romhar/absorbed.dat"
#define THREAD_PER_BLOCK 256 
#define NB_BLOCKS 256

char info[] = "\
Usage:\n\
    neutron-seq H Nb C_c C_s\n\
\n\
    H  : épaisseur de la plaque\n\
    Nb : nombre d'échantillons\n\
    C_c: composante absorbante\n\
    C_s: componente diffusante\n\
\n\
Exemple d'execution : \n\
    neutron-seq 1.0 500000000 0.5 0.5\n\
";


/*
 * notre gettimeofday()
 */
double my_gettimeofday(){
  struct timeval tmp_time;
  gettimeofday(&tmp_time, NULL);
  return tmp_time.tv_sec + (tmp_time.tv_usec * 1.0e-6L);
}

__global__ void setup_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x*gridDim.x;
    hiprand_init(16453, id, 0, &state[id]);
}

__global__ void neutron_calculus(hiprandState *state, float c, float c_c, float h, float* absorbed, int* result, int n, int* c_abs){
    int id = threadIdx.x + blockIdx.x*blockDim.x;
    int pos_ecrit;
    int pos_Thread = id;
    __shared__ int r[THREAD_PER_BLOCK];
    __shared__ int b[THREAD_PER_BLOCK];
    __shared__ int t[THREAD_PER_BLOCK];
    r[threadIdx.x] = 0;
    t[threadIdx.x] = 0;
    b[threadIdx.x] = 0;
    float L;
    float u;
    float d;
    float x;
    while(pos_Thread < n) {
	      d = 0.0;
        x = 0.0;
        while (1) {
				  u = hiprand_uniform (&state[id]);
				  L = -(1 / c) * log(u);
				  x = x + L * cos(d);
				  if (x < 0) {
						r[threadIdx.x] = r[threadIdx.x]+1;
						break;
								} else if (x >= h) {
						t[threadIdx.x] = t[threadIdx.x]+1;
						break;
							} else if ((u = hiprand_uniform (&state[id])) < c_c / c) {
		
					b[threadIdx.x] = b[threadIdx.x]+1;
					pos_ecrit = atomicAdd(c_abs, 1);
					absorbed[pos_ecrit] = x;
		
					break;
							} else {
					u = hiprand_uniform (&state[id]);
					d = u * M_PI;
							}
	    }
	pos_Thread = pos_Thread + gridDim.x*blockDim.x;
	}
	__syncthreads();
	int j = blockDim.x/2;
	while(j>0){
		if(threadIdx.x<j){
			r[threadIdx.x] += r[threadIdx.x + j];
			t[threadIdx.x] += t[threadIdx.x + j];
			b[threadIdx.x] += b[threadIdx.x + j];
		}
		j/=2;
		__syncthreads();
	}
	if(threadIdx.x==0){
		atomicAdd(result,r[0]);
		atomicAdd(result+1,t[0]);
		atomicAdd(result+2,b[0]);
	}
}

/*
 * main()
 */

int main(int argc, char *argv[]) {
  // La distance moyenne entre les interactions neutron/atome est 1/c. 
  // c_c et c_s sont les composantes absorbantes et diffusantes de c. 
  float c, c_c, c_s;
  // épaisseur de la plaque
  float h;
  // nombre d'échantillons
  int n;
  // nombre de neutrons refléchis, absorbés et transmis
  int* result = (int *) calloc(3, sizeof(int)); //r, t, b
  // chronometrage
  double start, finish;
  int j = 0; // compteurs 

  //perf files
  FILE *perf = fopen("../perform.txt", "a+");
  FILE *perf_gnuplot = fopen("../perform_gnuplot.txt", "a+");
  char str[512];
  //char tmp[128];

  if( argc == 1)
    fprintf( stderr, "%s\n", info);

  // valeurs par defaut
  h = 1.0;
  n = 500000000;
  c_c = 0.5;
  c_s = 0.5;

  // recuperation des parametres
  if (argc > 1)
    h = atof(argv[1]);
  if (argc > 2)
    n = atoi(argv[2]);
  if (argc > 3)
    c_c = atof(argv[3]);
  if (argc > 4)
    c_s = atof(argv[4]);
  c = c_c + c_s;

  // affichage des parametres pour verificatrion
  printf("Épaisseur de la plaque : %4.g\n", h);
  printf("Nombre d'échantillons  : %d\n", n);
  printf("C_c : %g\n", c_c);
  printf("C_s : %g\n", c_s);
  printf("Thread per block : %d\n",THREAD_PER_BLOCK);

	// INITIALISATION DES VARIABLES
  float *absorbed;
  absorbed = (float *) calloc(n, sizeof(float));
  int nb_thread = THREAD_PER_BLOCK;
  dim3 threadsParBloc(nb_thread,1,1);
  dim3 nbBlocks(NB_BLOCKS,1,1);
  float* absorbed_gpu;
  int* result_gpu;
  int* c_abs;
  hiprandState* d_state;

	// DEBUT CHRONO
	start = my_gettimeofday();

	// SETUP GPU (MALLOC)
  hipMalloc(&d_state, nb_thread*nbBlocks.x*sizeof(hiprandState));
  hipMalloc(&absorbed_gpu, n*sizeof(float));
  hipMalloc(&result_gpu, 3*sizeof(int));
  hipMalloc(&c_abs, sizeof(int));

	// INIT VARIABLE GPU (COPY, CALLOC)
  hipMemset(c_abs,0,sizeof(int));
	hipMemset(absorbed_gpu,0.0,n*sizeof(float));
  hipMemset(result_gpu, 0.0, 3*sizeof(int));

	// CALCUL
  setup_kernel<<<nbBlocks, threadsParBloc >>>(d_state); // RANDOM
  neutron_calculus<<<nbBlocks, threadsParBloc >>>(d_state, c, c_c, h, absorbed_gpu, result_gpu, n, c_abs); //CALCUL PARALLELE

	// RECUPERATION RESULT
  hipMemcpy(absorbed, absorbed_gpu, n*sizeof(float),hipMemcpyDeviceToHost);
  hipMemcpy(result, result_gpu, 3*sizeof(int),hipMemcpyDeviceToHost);

	// FIN CHRONO
  finish = my_gettimeofday();
  float* var_test_0 = (float*)calloc(500000000, sizeof(float));
  float* var_test_1 = (float*)malloc(500000000*sizeof(float));
  float* var_cuda0, *var_cuda1;
  hipMalloc((void**)&var_cuda0,500000000*sizeof(float));
  hipMalloc((void**)&var_cuda1, 500000000*sizeof(float));
  srand((unsigned int)time(NULL));
  for(int i=0;i<500000000;i++)	var_test_1[i]=(float)rand()/(float)(RAND_MAX);

  start = my_gettimeofday();
  hipMemcpy(var_cuda0,var_test_0,500000000*sizeof(float),hipMemcpyHostToDevice);
  finish = my_gettimeofday();
  printf("\n Temps pour envoyer des 0 : %.8g sec\n", finish - start);

  start = my_gettimeofday();
  hipMemcpy(var_cuda1,var_test_1,500000000*sizeof(float),hipMemcpyHostToDevice);
  finish = my_gettimeofday();  
  printf("\n Temps pour envoyer des rand : %.8g sec\n", finish - start);

  start = my_gettimeofday();
  hipMemcpy(var_test_0,var_cuda0,500000000*sizeof(float),hipMemcpyDeviceToHost);
  finish = my_gettimeofday();
  printf("\n Temps pour recevoir des 0 : %.8g sec\n", finish - start);

  start = my_gettimeofday();
  hipMemcpy(var_test_1, var_cuda1,500000000*sizeof(float),hipMemcpyDeviceToHost);
  finish = my_gettimeofday();
  printf("\n Temps pour recevoir des rand : %.8g sec\n", finish - start);

  printf("\nTemps total de calcul: %.8g sec\n", finish - start);
  sprintf(str,"***************Cuda N:%d ***************\n\
Nb_thread:%d , Nb_Blocs:%d \n\
#Temps total de calcul : %.8g seconde(s)\n\n"
            ,n,THREAD_PER_BLOCK,NB_BLOCKS,finish-start);

	fwrite(str,sizeof(char),strlen(str),perf);
	sprintf(str,"%d %.8g %d %d \n",n,finish-start,THREAD_PER_BLOCK,NB_BLOCKS);
	fwrite(str,sizeof(char),strlen(str),perf_gnuplot);

  int r = result[0];
  int t = result[1];
  int b = result[2];
  printf("\nPourcentage des neutrons refléchis : %4.2g\n", (float) r / (float) n);
  printf("Pourcentage des neutrons absorbés : %4.2g\n", (float) b / (float) n);
  printf("Pourcentage des neutrons transmis : %4.2g\n", (float) t / (float) n);
  printf("Nombre de neutrons traites : %d\n", r+b+t);
  printf("\nTemps total de calcul: %.8g sec\n", finish - start);
  printf("Millions de neutrons /s: %.2g\n", (double) n / ((finish - start)*1e6));

  // ouverture du fichier pour ecrire les positions des neutrons absorbés
  FILE *f_handle = fopen(OUTPUT_FILE, "w+");
  if (!f_handle) {
    fprintf(stderr, "Cannot open " OUTPUT_FILE "\n");
    exit(EXIT_FAILURE);
  }

  for (j = 0; j < b; j++)
    fprintf(f_handle, "%f\n", absorbed[j]);

  // fermeture du fichier
  fclose(f_handle);
  printf("Result written in " OUTPUT_FILE "\n\n "); 

  free(absorbed);
  fclose(perf);
  fclose(perf_gnuplot);
  fclose(f_handle);

  return EXIT_SUCCESS;
}

